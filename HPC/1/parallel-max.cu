//parallel max using cuda//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h> // for rand()
#include <errno.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_math_constants.h>

#define N 2*2*2

/* host buffer */
float *data;
/* device buffers */
float *dSrc, *dDst;

void check_error(hipError_t error, const char *message) {
    if (error != hipSuccess) {
        fprintf(stderr, "%s (%s)\n", message,
            hipGetErrorString(error));
        if (dSrc)
            hipFree(dSrc);
        if (dDst)
            hipFree(dDst);
        exit(1);
    }
}

#define WARP_SIZE 32
#define BLOCK_SIZE (12*WARP_SIZE)

__global__ void findmax(float *dDst, const float *dSrc, uint dim)
{
    __shared__ float cache[BLOCK_SIZE];					

    uint gix = threadIdx.x + blockDim.x*blockIdx.x;

#define tid threadIdx.x

    float acc = HIP_NAN_F;		

    while (gix < dim) {				
        acc = fmax(acc, dSrc[gix]);
        gix += blockDim.x*gridDim.x;
    }

    cache[tid] = acc;				
    uint active = blockDim.x >> 1;	

    do {
        __syncthreads();			
        if (tid < active)
            cache[tid] = fmax(cache[tid], cache[tid+active]);			
        active >>= 1;
    } while (active > 0);

    if (tid == 0)					
        dDst[blockIdx.x] = cache[0];
}

int main(int argc, char **argv) {
    data = (float*) calloc(N, sizeof(float));
    size_t data_size = N * sizeof(float);				
    float max = nan(""), d_max = nan("");				

    for (size_t i = 0; i < N; ++i) {
        data[i] = i; 									
        max = fmax(max, data[i]);						
    }
    printf("%u elements generated, max %g, data size %zu (%zuMB)\n",
            N, max, data_size, data_size>>20);

    hipError_t err;

    err = hipMalloc(&dSrc, data_size);
    check_error(err, "allocating array");

    err = hipMemcpy(dSrc, data, data_size, hipMemcpyHostToDevice);		
    check_error(err, "copy UP");

    uint nblocks = 8;

    err = hipMalloc(&dDst, nblocks*sizeof(*dDst));
    check_error(err, "allocating Dst array");

    hipEvent_t start, stop;
    float runtime;
    hipEventCreate(&start);				
    hipEventCreate(&stop);					

    hipEventRecord(start, 0);				
    findmax<<<nblocks,BLOCK_SIZE>>>(dDst, dSrc, N);		
    findmax<<<1,BLOCK_SIZE>>>(dDst, dDst, nblocks);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);				
    hipEventElapsedTime(&runtime, start, stop);		

    /* Giga-elements per second */
    printf("%u elements processed in %gms: %gGE/s\n",
        N, runtime, (N/runtime)/(1000000));

    /* Actual bandwith in GB/s */
    uint total_els = N + nblocks;
    float sizeMB = float(total_els)*sizeof(float)/(1024*1024);
    printf("Bandwidth: %u elements (%gMB) read in two steps. "
        "Runtime: %gms (%gGB/s)\n",
        total_els, sizeMB, runtime, sizeMB/runtime);

    err = hipMemcpy(&d_max, dDst, sizeof(d_max), hipMemcpyDeviceToHost);		
    check_error(err, "copy DOWN");

    hipFree(dSrc); dSrc = NULL;
    hipFree(dDst); dDst = NULL;
    free(data);

    printf("Parallel max: %g vs %g\n", d_max, max);
}

