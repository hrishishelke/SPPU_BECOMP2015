#include "hip/hip_runtime.h"

//Parallel sum in cuda.


#include "hip/hip_runtime.h"
#include ""
#include<bits/stdc++.h>
#include <iostream>
#include <numeric>

using namespace std;

#define BLOCK_SIZE 4

__global__ void sum(int* input)  		/
{
	const int tid = threadIdx.x;
	int step_size = 1;
	int number_of_threads = blockDim.x;  
	

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) 
		{     
			const int fst = tid * step_size * 2;    
			const int snd = fst + step_size;		
			input[fst] += input[snd];				
					
		}

		step_size <<= 1; 			
		number_of_threads >>= 1;		
	}
	
}

int main()
{
	int count=0;
	cout << "Enter the number of elements:\n" << endl;
	cin>>count;

	const int size = count * sizeof(int);
	
	int h[count];
	cout << "Enter the elements:\n" << endl;
	for(int i=0;i<count;i++)
	{
		cout << "\n";
		cin >> h[i];
	}
	

	int* d;							
       
	
	hipMalloc(&d, size);			
	
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);		
	sum <<<1, count / 2 >>>(d);							
	
	int result;
	
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);		

	cout << "Sum is " << result << endl;

	getchar();

	hipFree(d);					


	return 0;
}
